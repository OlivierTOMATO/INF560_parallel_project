/**
 * APPROXIMATE PATTERN MATCHING
 *
 * INF560
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>

#define APM_DEBUG 0

char *
read_input_file(char *filename, int *size)
{
    char *buf;
    off_t fsize;
    int fd = 0;
    int n_bytes = 1;

    /* Open the text file */
    fd = open(filename, O_RDONLY);
    if (fd == -1)
    {
        fprintf(stderr, "Unable to open the text file <%s>\n", filename);
        return NULL;
    }

    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    if (fsize == -1)
    {
        fprintf(stderr, "Unable to lseek to the end\n");
        return NULL;
    }

#if APM_DEBUG
    printf("File length: %lld\n", fsize);
#endif

    /* Go back to the beginning of the input file */
    if (lseek(fd, 0, SEEK_SET) == -1)
    {
        fprintf(stderr, "Unable to lseek to start\n");
        return NULL;
    }

    /* Allocate data to copy the target text */
    buf = (char *)malloc(fsize * sizeof(char));
    if (buf == NULL)
    {
        fprintf(stderr, "Unable to allocate %lld byte(s) for main array\n",
                fsize);
        return NULL;
    }

    n_bytes = read(fd, buf, fsize);
    if (n_bytes != fsize)
    {
        fprintf(stderr,
                "Unable to copy %lld byte(s) from text file (%d byte(s) copied)\n",
                fsize, n_bytes);
        return NULL;
    }

#if APM_DEBUG
    printf("Number of read bytes: %d\n", n_bytes);
#endif

    *size = n_bytes;

    close(fd);

    return buf;
}

// #define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))
int __device__ min3(int a, int b, int c)
{
    return ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)));
}

int __device__ levenshtein_cuda(char *s1, char *s2, int len, int *column)
{
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++)
    {
        column[y] = y;
    }
    for (x = 1; x <= len; x++)
    {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++)
        {
            olddiag = column[y];
            column[y] = min3(
                column[y] + 1,
                column[y - 1] + 1,
                lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    return (column[len]);
}

void __global__ matches_kernel(char *d_buf, char *d_pattern, int *d_num, int size_pattern, int start, int end, int n_bytes, int approx_factor)
{

    /* Traverse the input data up to the end of the file */
    int j = blockIdx.x * blockDim.x + threadIdx.x + start;
    int stride = blockDim.x * gridDim.x;
    int distance = 0;
    int size;
    int num_local = 0;

    size = size_pattern;
    int *columns = (int *)malloc((size_pattern + 1) * sizeof(int));
    while (j < end)
    {
        if (n_bytes - j < size_pattern)
        {
            size = n_bytes - j;
        }

        distance = levenshtein_cuda(d_pattern, &d_buf[j], size, columns);
        if (distance <= approx_factor)
        {
            atomicAdd(&d_num[0], 1);
        }
        j += stride;
    }
    free(columns);
}

int main(int argc, char **argv)
{
    char **pattern;
    char *filename;
    int approx_factor = 0;
    int nb_patterns = 0;
    int i, j;
    char *buf;
    struct timeval t1, t2;
    double duration;
    int n_bytes;
    int *n_matches;

    /* Check number of arguments */
    if (argc < 4)
    {
        printf("Usage: %s approximation_factor "
               "dna_database pattern1 pattern2 ...\n",
               argv[0]);
        return 1;
    }

    /* Get the distance factor */
    approx_factor = atoi(argv[1]);

    /* Grab the filename containing the target text */
    filename = argv[2];

    /* Get the number of patterns that the user wants to search for */
    nb_patterns = argc - 3;

    /* Fill the pattern array */
    pattern = (char **)malloc(nb_patterns * sizeof(char *));
    if (pattern == NULL)
    {
        fprintf(stderr,
                "Unable to allocate array of pattern of size %d\n",
                nb_patterns);
        return 1;
    }

    /* Grab the patterns */
    for (i = 0; i < nb_patterns; i++)
    {
        int l;

        l = strlen(argv[i + 3]);
        if (l <= 0)
        {
            fprintf(stderr, "Error while parsing argument %d\n", i + 3);
            return 1;
        }

        pattern[i] = (char *)malloc((l + 1) * sizeof(char));
        if (pattern[i] == NULL)
        {
            fprintf(stderr, "Unable to allocate string of size %d\n", l);
            return 1;
        }

        strncpy(pattern[i], argv[i + 3], (l + 1));
    }

    printf("Approximate Pattern Mathing: "
           "looking for %d pattern(s) in file %s w/ distance of %d\n",
           nb_patterns, filename, approx_factor);

    buf = read_input_file(filename, &n_bytes);
    if (buf == NULL)
    {
        return 1;
    }

    /* Allocate the array of matches */
    n_matches = (int *)malloc(nb_patterns * sizeof(int));
    if (n_matches == NULL)
    {
        fprintf(stderr, "Error: unable to allocate memory for %ldB\n",
                nb_patterns * sizeof(int));
        return 1;
    }

    /*****
     * BEGIN MAIN LOOP
     ******/

    /* Timer start */
    gettimeofday(&t1, NULL);


    
    int *local_n_matches = (int *)malloc(nb_patterns * sizeof(int));

    char *d_buf;
    int *d_num;
    char *d_pattern;
    hipMalloc((void **)&d_buf, n_bytes * sizeof(char));
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);
    /* Check each pattern one by one */
    for (i = 0; i < nb_patterns; i++)
    { 
        int size_pattern = strlen(pattern[i]);

        hipMalloc((void **)&d_num, sizeof(int));
        hipMalloc((void **)&d_pattern, size_pattern * sizeof(char));
        hipMemcpy(d_pattern, pattern[i],  size_pattern * sizeof(char), hipMemcpyHostToDevice);
       
        /* Initialize the number of matches to 0 */
        n_matches[i] = 0;
        /* Traverse the input data up to the end of the file */
        matches_kernel<<<4, 256>>>(d_buf, d_pattern, d_num, size_pattern, 0, n_bytes, n_bytes, approx_factor);

        hipMemcpy(&local_n_matches[i], d_num, sizeof(int), hipMemcpyDeviceToHost);
        n_matches[i] += local_n_matches[i];

        hipFree(d_pattern);
        hipFree(d_num);
    }

    /* Timer stop */
    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);

    printf("APM done in %lf s\n", duration);

    /*****
     * END MAIN LOOP
     ******/

    for (i = 0; i < nb_patterns; i++)
    {
        printf("Number of matches for pattern <%s>: %d\n",
               pattern[i], n_matches[i]);
    }

    return 0;
}
